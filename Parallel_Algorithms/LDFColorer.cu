#include "hip/hip_runtime.h"
#include "graph\graph.h"
#include "graph\graph_d.h"
#include "graph\coloring.h"
#include "utils\common.h"

#define THREADxBLOCK 128

using namespace std;

__global__ void colorer(Coloring * col, GraphStruct *str){
	int n = str->nodeSize;

	for (int i = threadIdx.x+blockIdx.x*blockDim.x; i < n; i += blockDim.x*gridDim.x) {
		bool flag = true; // vera sse il nodo ha peso locale massimo

		// ignora i nodi già colorati
		if ((col->coloring[i] != -1)) continue;

		int iWeight = str->weights[i];
		bool* forbidden;
		hipMalloc((void**) &forbidden, n * sizeof(bool));
		memset(forbidden, false, n);

		// guarda i pesi del vicinato
		uint offset = str->cumDegs[i];
		uint deg = str->cumDegs[i + 1] - str->cumDegs[i];

		for (uint j = 0; j < deg; j++) {
			uint neighID = str->neighs[offset + j];
			int jColor = col->coloring[neighID];

			if (jColor != -1 || i == neighID) {
					forbidden[jColor] = true;
					continue;
      }

			int jWeight = str->weights[neighID];
			uint neighDeg = str->cumDegs[neighID + 1] - str->cumDegs[neighID];
			if ((deg < neighDeg) || ((deg == neighDeg) && (iWeight < jWeight))) flag = false;
		}

		// colora solo se sei il nodo di peso massimo
		if (flag){

			for(int c = 0; c < n; c++){
					if(!forbidden[c]) { 
							col->coloring[i] = c;
							break;
					}
      }
			free(forbidden);
		}
}

void FYshuffle(int * weights, uint n){
    for(int i = 0; i < n; i++){
        int swapIdx = (rand() % (n - i)) + i;
        int tmp = weights[i];
				weights[i] = weights[swapIdx];
				weights[swapIdx] = tmp;
    }
}

Coloring* graphColoring(GraphStruct *str){
	int n = str->nodeSize;
	Coloring* col;
	CHECK(hipMallocManaged(&col, sizeof(Coloring)));
	CHECK(hipMallocManaged(&(col->coloring), n * sizeof(int)));
	memset(col->coloring, -1 ,n * sizeof(int));

	dim3 threads ( THREADxBLOCK);
	dim3 blocks ((str->nodeSize + threads.x - 1) / threads.x, 1, 1 );

	for (int i = 0; i < n; i++){
				str->weights[i] = i;
		}

	FYshuffle(str->weights, n);

	printf("Pesi: ");
	for(int i = 0; i < n; i++){
		printf("%d ", str->weights[i]);
	}
	printf("\n");

	//print_d <<< 1, 1 >>> (str, true);

	bool flag=true;
	while(flag){
		colorer<<<blocks, threads>>>(col, str);
		hipDeviceSynchronize();
		flag=false;
		for(int i=0; i<n; i++){
			if(col->coloring[i]==-1){
				flag=true;
			}
		}
	}
  return col;
}