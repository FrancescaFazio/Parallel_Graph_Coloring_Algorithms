#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>
#include "graph\graph.h"
#include "graph\graph_d.h"
#include "graph\coloring.h"
#include "utils\common.h"
#include <thrust/sequence.h>
#include <thrust/shuffle.h>
#include <thrust/random.h>
#include <thrust/count.h>

#define THREADxBLOCK 128

using namespace std;

__global__ void findIS(Coloring * col, GraphStruct *str, bool * unvisitedNodes, int * currentMIS){
	int n = str->nodeSize;

	for (int i = threadIdx.x+blockIdx.x*blockDim.x; i < n; i += blockDim.x*gridDim.x) {
		bool flag = true; // vera sse il nodo ha peso locale massimo

		// ignora i nodi già colorati
		if ((col->coloring[i] != -1) || (currentMIS[i] == 1) || (currentMIS[i] == -1)) continue;

		int iWeight = str->weights[i];

		// guarda i pesi del vicinato
		uint offset = str->cumDegs[i];
		uint deg = str->cumDegs[i + 1] - str->cumDegs[i];

		for (uint j = 0; j < deg; j++) {
			uint neighID = str->neighs[offset + j];
			// ignora i vicini già colorati (e te stesso)
			int jColor = col->coloring[neighID];
			if ((jColor != -1) || currentMIS[neighID] == -1 || (i == neighID)) continue;
			int jWeight = str->weights[neighID];
			if (iWeight <= jWeight) flag = false;
		}
		// colora solo se sei il nodo di peso massimo

		if (flag){
			currentMIS[i] = 1;
			for (uint j = 0; j < deg; j++) {
				uint neighID = str->neighs[offset + j];
				if (currentMIS[neighID] == -1 || (i == neighID)) continue;
				currentMIS[neighID] = -1;
			}
		}else if (currentMIS[i] == 0){
			* unvisitedNodes = true;
		}
	}
}

__global__ void colorer(Coloring * col, GraphStruct *str, int * currentMIS){
	int n = str->nodeSize;

	for (int i = threadIdx.x+blockIdx.x*blockDim.x; i < n; i += blockDim.x*gridDim.x) {
		if(currentMIS[i] == 1){
			col->coloring[i] = col->numOfColors;

		}
		else if(currentMIS[i] == -1 && col->coloring[i] == -1) col->uncoloredNodes = true;
	}
}

Coloring* graphColoring(GraphStruct *str){
	int n = str->nodeSize;
	printf("%d\n", n);

	// Creazione coloratura CPU e GPU
	Coloring * col_h;
	Coloring * col_d;
	int * coloring_d;
	bool * unvisitedNodes_h;
	bool * unvisitedNodes_d;
	int * currentMIS;

	// CPU
	col_h = (Coloring *) malloc(sizeof(Coloring));

	col_h->coloring = (int *) malloc(n * sizeof(int));
	thrust::fill(col_h->coloring, col_h->coloring + n, -1);

	unvisitedNodes_h = (bool *)malloc(sizeof(bool));

	col_h->numOfColors = 0;
	col_h->uncoloredNodes = true;
	* unvisitedNodes_h = true;

	// GPU
	gpuErrchk(hipMalloc((void **) &col_d, sizeof(Coloring)));
	gpuErrchk(hipMalloc((void **) &coloring_d, n * sizeof(int)));
	gpuErrchk(hipMalloc((void **) &unvisitedNodes_d, sizeof(bool)));
	gpuErrchk(hipMalloc((void **) &currentMIS, n * sizeof(int)));

	gpuErrchk(hipMemcpy(coloring_d, col_h->coloring, n * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(&(col_d->coloring), &coloring_d, sizeof(col_d->coloring), hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(unvisitedNodes_d, false, sizeof(bool)));
	gpuErrchk(hipMemset(&(col_d->numOfColors), 0, sizeof(uint)));
	gpuErrchk(hipMemset(&(col_d->uncoloredNodes), false, sizeof(bool)));
	gpuErrchk(hipMemset(currentMIS, 0, n * sizeof(int)));

    // Generazione pesi
    thrust::sequence(str->weights, str->weights + n);
    thrust::default_random_engine g;
    thrust::shuffle(str->weights, str->weights + n, g);

	dim3 threads (THREADxBLOCK);
	dim3 blocks ((str->nodeSize + threads.x - 1) / threads.x, 1, 1 );

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord(start);

	while(col_h->uncoloredNodes){
		col_h->numOfColors++;
		col_h->uncoloredNodes = false;

		// Aggiorno coloring GPU
		gpuErrchk(hipMemcpy(&(col_d->uncoloredNodes), &(col_h->uncoloredNodes), sizeof(bool), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(&(col_d->numOfColors), &(col_h->numOfColors), sizeof(uint), hipMemcpyHostToDevice));

		while(*unvisitedNodes_h){
			gpuErrchk(hipMemset(unvisitedNodes_d, false, sizeof(bool)));

			findIS<<<blocks, threads>>>(col_d, str, unvisitedNodes_d, currentMIS);
        	gpuErrchk(hipPeekAtLastError())
			gpuErrchk(hipDeviceSynchronize());

			gpuErrchk(hipMemcpy(unvisitedNodes_h, unvisitedNodes_d, sizeof(bool), hipMemcpyDeviceToHost));
		}

		// Coloro il MIS trovato
		colorer<<<blocks, threads>>>(col_d, str, currentMIS);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		// Resetto il MIS e copio in CPU uncoloredNodes
		gpuErrchk(hipMemset(currentMIS, 0, n * sizeof(int)));
		gpuErrchk(hipMemcpy(&(col_h->uncoloredNodes), &(col_d->uncoloredNodes), sizeof(bool), hipMemcpyDeviceToHost));
		*unvisitedNodes_h = true;
	}

    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tempo solo kernel %f ms\n", milliseconds);

	gpuErrchk(hipMemcpy(col_h->coloring, coloring_d, n * sizeof(int), hipMemcpyDeviceToHost));

	return col_h;
}
