#include <iostream>
#include "graph\graph.h"
#include "graph\coloring.h"
#include "utils\common.h"

#define THREADxBLOCK 128

int main(void) {
    //unsigned int n = 50515;		
    //float prob = 0;				    
    //std::default_random_engine eng{0}; 

    srand(time(0));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // new graph with n nodes
    Graph graph("facebook_clean_data/com-amazon.ungraph.txt", 1);
    //Graph graph(n, 1);
    //graph.randGraph(prob, eng);

    // get the graph struct
    GraphStruct *str = graph.getStruct();

    hipEventRecord(start);

    Coloring* col = graphColoring(str);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Stampo in millisecondi quanto tempo ci ha messo a colorare il grafo.
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f ms\n", milliseconds);

    int maxColor = 0;
    for(int i = 0; i < str->nodeSize; i++){
        if(maxColor < col->coloring[i]) maxColor = col->coloring[i];
        //printf("%d ", col->coloring[i]);
    }
    printf("\nColore massimo: %d", maxColor+1);
    //printColoring(col, str, 1);

    return EXIT_SUCCESS;
}