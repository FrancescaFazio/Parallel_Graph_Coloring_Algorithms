#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <stdlib.h>
#include <hiprand.h>
#include "graph\graph.h"
#include "graph\graph_d.h"
#include "graph\coloring.h"
#include "utils\common.h"
#include <thrust/sequence.h>
#include <thrust/shuffle.h>
#include <thrust/random.h>
#include <thrust/count.h>

#define THREADxBLOCK 128
#define GRAPH_DIM 1000

using namespace std;

__device__ int random;

__global__ void setDegrees(GraphStruct *str, uint * k, uint * weight, bool * flag, uint * visitedNodes){
    uint n = str->nodeSize;

    for (int idx = threadIdx.x+blockIdx.x*blockDim.x; idx < n; idx += blockDim.x*gridDim.x){
        uint offset = str->cumDegs[idx];
        uint originalDeg = str->cumDegs[idx + 1] - str->cumDegs[idx];
        uint inducedDeg = 0;

        for (uint i = 0; i < originalDeg; i++){
            uint neighID = str->neighs[offset + i];
            if(str->weights[neighID] == -1) inducedDeg += 1;
        }

        if (inducedDeg <= * k && str->weights[idx] == -1){
            str->weights[idx] = * weight;
            * flag = true;
            * visitedNodes += 1;
        } 
    }
}

void initDegrees(GraphStruct *str){
    // CPU
    uint n = str->nodeSize;

    dim3 threads ( THREADxBLOCK);
    dim3 blocks ((n + threads.x - 1) / threads.x, 1, 1 );

    int nodesLeft = n;
    bool nodeSeen_h;
    uint visitedNodes_h;
    uint degree, weight;
    degree = 1; weight = 1;

    nodeSeen_h = true;
    visitedNodes_h = 0;

    // GPU
    bool * nodeSeen_d;
    uint * degree_d;
    uint * weigth_d;
    uint * visitedNodes_d;

    gpuErrchk(hipMalloc((void **) &nodeSeen_d, sizeof(bool)));
    gpuErrchk(hipMalloc((void **) &degree_d, sizeof(uint)));
    gpuErrchk(hipMalloc((void **) &weigth_d, sizeof(uint)));
    gpuErrchk(hipMalloc((void **) &visitedNodes_d, sizeof(uint)));

    gpuErrchk(hipMemset(nodeSeen_d, true, sizeof(bool)));
    gpuErrchk(hipMemset(degree_d, 1, sizeof(uint)));
    gpuErrchk(hipMemset(weigth_d, 1, sizeof(uint)));
    gpuErrchk(hipMemset(visitedNodes_d, 0, sizeof(uint)));

    while(nodesLeft > 0){
        while(nodeSeen_h){
            // Aggiorno flag
            gpuErrchk(hipMemset(nodeSeen_d, false, sizeof(bool)));

            setDegrees <<< blocks, threads >>> (str, degree_d, weigth_d, nodeSeen_d, visitedNodes_d);
            gpuErrchk(hipPeekAtLastError())
		    gpuErrchk(hipDeviceSynchronize());

            weight += 1;

            gpuErrchk(hipMemcpy(&nodeSeen_h, nodeSeen_d, sizeof(bool), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(&visitedNodes_h, visitedNodes_d, sizeof(uint), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(weigth_d, &weight, sizeof(uint), hipMemcpyHostToDevice));

            nodesLeft -= visitedNodes_h;
        }
        nodeSeen_h = true;
        degree += 1; 
        gpuErrchk(hipMemcpy(degree_d, &degree, sizeof(uint), hipMemcpyHostToDevice));
    }
}

__global__ void findCandidates (Coloring* col, GraphStruct *str, bool * currentIS, bool * usedColors) {
	int n = str->nodeSize;

	for (int i = threadIdx.x+blockIdx.x*blockDim.x; i < n; i += blockDim.x*gridDim.x) {
		bool flag = true; // vera sse il nodo ha peso locale massimo
        
		// ignora i nodi già colorati
		if ((col->coloring[i] != -1)) continue;

		int iWeight = str->weights[i];

		// guarda i pesi del vicinato
		uint offset = str->cumDegs[i];
		int deg = str->cumDegs[i + 1] - str->cumDegs[i];

		for (uint j = 0; j < deg; j++) {
			uint neighID = str->neighs[offset + j];

			// ignora i vicini già colorati (e te stesso)
			int jColor = col->coloring[neighID];

            if(jColor != -1 && jColor < deg){
                usedColors[offset + jColor] = true;
                continue;
            }
            if(jColor != -1 || i == neighID) continue;
            
			int jWeight = str->weights[neighID];
            
            if (iWeight < jWeight) flag = false;
            else if (iWeight == jWeight){
                int iRandom = (i + random) % n;
                int jRandom = (neighID + random) % n;
                if(iRandom < jRandom) flag = false;
            }
		}
           
		// colora solo se sei il nodo di peso massimo
		if (flag) currentIS[i] = true;
    }
}

__global__ void colorer(Coloring * col, GraphStruct *str, bool * currentIS, bool * usedColors){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    uint n = str->nodeSize;

    if (i >= n) return;

    if (currentIS[i] == 1 && col->coloring[i] == -1){
        int color = 1;
        uint offset = str->cumDegs[i];

        // Cerca il primo colore libero per questo nodo
        while (usedColors[offset + color]) color++;
        
        // Assegna il primo colore libero al nodo corrente
        col->coloring[i] = color;
        //currentIS[i] = false;
        //col->usedColors[color] = true; 
    }else if(currentIS[i] == 0 && col->coloring[i] == -1){
        col->uncoloredNodes = true;
    }
}

Coloring* graphColoring(GraphStruct *str) {
	int n = str->nodeSize;
    int r = rand();
    bool * currentIS;
	printf("%d ",n);

	// Creazione coloratura CPU e GPU
	Coloring * col_h;
	Coloring * col_d;
	int * coloring_d;
    bool * usedColors_d;

    // Generazione pesi
    thrust::sequence(str->weights, str->weights + n);
    thrust::default_random_engine g;
    thrust::shuffle(str->weights, str->weights + n, g);

	// CPU
	col_h = (Coloring *) malloc(sizeof(Coloring));

	col_h->coloring = (int *) malloc(n * sizeof(int));
	thrust::fill(col_h->coloring, col_h->coloring + n, -1);


	col_h->numOfColors = 0;
	col_h->uncoloredNodes = true;

	// GPU
	gpuErrchk(hipMalloc((void **) &col_d, sizeof(Coloring)));
	gpuErrchk(hipMalloc((void **) &coloring_d, n * sizeof(int)));
    gpuErrchk(hipMalloc((void **) &currentIS, n * sizeof(bool)));
    gpuErrchk(hipMalloc((void **) &usedColors_d, str->edgeSize * sizeof(bool)));

	gpuErrchk(hipMemcpy(coloring_d, col_h->coloring, n * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(&(col_d->coloring), &coloring_d, sizeof(col_d->coloring), hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(&(col_d->numOfColors), 0, sizeof(uint)));
	gpuErrchk(hipMemset(&(col_d->uncoloredNodes), false, sizeof(bool)));
    gpuErrchk(hipMemset(currentIS, false, n * sizeof(bool)));
	gpuErrchk(hipMemset(usedColors_d, false, str->edgeSize * sizeof(bool)));

	dim3 threads (THREADxBLOCK);
	dim3 blocks ((str->nodeSize + threads.x - 1) / threads.x, 1, 1 );

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord(start);

	while(col_h->uncoloredNodes){
		col_h->uncoloredNodes = false;
		col_h->numOfColors++;

		// Aggiorno coloring GPU
		gpuErrchk(hipMemcpy(&(col_d->uncoloredNodes), &(col_h->uncoloredNodes), sizeof(bool), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(&(col_d->numOfColors), &(col_h->numOfColors), sizeof(uint), hipMemcpyHostToDevice));

		findCandidates<<<blocks, threads>>>(col_d, str, currentIS, usedColors_d);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        colorer<<<blocks, threads>>>(col_d, str, currentIS, usedColors_d);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

		// Aggiorno uncoloredNodes lato CPU e resetto currentIS
		gpuErrchk(hipMemcpy(&(col_h->uncoloredNodes), &(col_d->uncoloredNodes), sizeof(bool), hipMemcpyDeviceToHost));
        //gpuErrchk(hipMemset(currentIS, false, n * sizeof(bool)));
	}

    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tempo solo kernel %f ms\n", milliseconds);

	// Copio il risultato da CPU a GPU
	gpuErrchk(hipMemcpy(col_h->coloring, coloring_d, n * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&(col_h->numOfColors), &(col_d->numOfColors), sizeof(uint), hipMemcpyDeviceToHost));

    return col_h;
}


			