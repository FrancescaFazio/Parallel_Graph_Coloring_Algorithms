#include "hip/hip_runtime.h"
#include "graph\graph.h"
#include "graph\graph_d.h"
#include "graph\coloring.h"
#include "utils\common.h"

#define THREADxBLOCK 128

using namespace std;

__global__ void colorer(Coloring * col, GraphStruct *str){
	int n = str->nodeSize;

	for (int i = threadIdx.x+blockIdx.x*blockDim.x; i < n; i += blockDim.x*gridDim.x) {
		bool flag = true; // vera sse il nodo ha peso locale massimo

		// ignora i nodi già colorati
		if ((col->coloring[i] != -1)) continue;

		int iWeight = str->weights[i];

		// guarda i pesi del vicinato
		uint offset = str->cumDegs[i];
		uint deg = str->cumDegs[i + 1] - str->cumDegs[i];

		for (uint j = 0; j < deg; j++) {
			uint neighID = str->neighs[offset + j];
			// ignora i vicini già colorati (e te stesso)
			int jColor = col->coloring[neighID];
			if (((jColor != -1) && (jColor != col->numOfColors)) || (i == neighID)) continue;
			int jWeight = str->weights[neighID];
			if (iWeight <= jWeight) flag = false;
		}
		// colora solo se sei il nodo di peso massimo
		if (flag) col->coloring[i] = col->numOfColors;
  	}
}

void FYshuffle(int * weights, uint n){
    for(int i = 0; i < n; i++){
        int swapIdx = (rand() % (n - i)) + i;
        int tmp = weights[i];
				weights[i] = weights[swapIdx];
				weights[swapIdx] = tmp;
    }
}

Coloring* graphColoring(GraphStruct *str){
	int n = str->nodeSize;
	Coloring* col;
	CHECK(hipMallocManaged(&col, sizeof(Coloring)));
	col->uncoloredNodes = true;

    CHECK(hipMallocManaged(&(col->coloring), n * sizeof(int)));
	memset(col->coloring, -1 ,n * sizeof(int));
	// allocate space on the GPU for the random states

	dim3 threads ( THREADxBLOCK);
	dim3 blocks ((str->nodeSize + threads.x - 1) / threads.x, 1, 1 );

	for (int i = 0; i < n; i++){
        str->weights[i] = i;
    }
    FYshuffle(str->weights, n);

	bool flag = false;
	for(int c = 0; c < n; c++){
		col->numOfColors = c;
		colorer<<<blocks, threads>>>(col, str);
		hipDeviceSynchronize();

		for(int i=0; i<n; i++){
				if(col->coloring[i] == -1){
						flag = true;
				}
		}

		if(!flag) break;
		else flag = false;

	}

    return col;
}
