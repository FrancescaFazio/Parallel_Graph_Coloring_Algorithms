#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "graph\graph.h"
#include "graph\graph_d.h"
#include "graph\coloring.h"
#include "utils\common.h"
#include <thrust/sequence.h>
#include <thrust/shuffle.h>
#include <thrust/random.h>
#include <thrust/count.h>

#define THREADxBLOCK 128

using namespace std;

__global__ void colorer(Coloring * col, GraphStruct * str){
    uint n = str->nodeSize;

    for (int i = threadIdx.x+blockIdx.x*blockDim.x; i < n; i += blockDim.x*gridDim.x) {
        bool flag = true; // vera sse il nodo ha peso locale massimo

        // ignora i nodi già colorati
        if ((col->coloring[i] != -1)) return;

        int iWeight = str->weights[i];

        // guarda i pesi del vicinato
        uint offset = str->cumDegs[i];
        uint deg = str->cumDegs[i + 1] - str->cumDegs[i];
        
        for (uint j = 0; j < deg; j++) {
            uint neighID = str->neighs[offset + j];
            // ignora i vicini già colorati (e te stesso)
            int jColor = col->coloring[neighID];
            if (((jColor != -1) && (jColor != col->numOfColors)) || (i == neighID)) continue;
            int jWeight = str->weights[neighID];
            if (iWeight <= jWeight) flag = false;
        }

        // colora solo se sei il nodo di peso massimo
        if (flag) col->coloring[i] = col->numOfColors;
        else col->uncoloredNodes = true;
    }
}

Coloring* graphColoring(GraphStruct *str) {
	int n = str->nodeSize;
	printf("%d ",n);

	// Creazione coloratura CPU e GPU
	Coloring * col_h;
	Coloring * col_d;
	int * coloring_d;

    // Generazione pesi
    thrust::sequence(str->weights, str->weights + n);
    thrust::default_random_engine g;
    thrust::shuffle(str->weights, str->weights + n, g);

	// CPU
	col_h = (Coloring *) malloc(sizeof(Coloring));

	col_h->coloring = (int *) malloc(n * sizeof(int));
	thrust::fill(col_h->coloring, col_h->coloring + n, -1);

	col_h->numOfColors = 0;
	col_h->uncoloredNodes = true;

	// GPU
	gpuErrchk(hipMalloc((void **) &col_d, sizeof(Coloring)));
	gpuErrchk(hipMalloc((void **) &coloring_d, n * sizeof(int)));

	gpuErrchk(hipMemcpy(coloring_d, col_h->coloring, n * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(&(col_d->coloring), &coloring_d, sizeof(col_d->coloring), hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(&(col_d->numOfColors), 0, sizeof(uint)));
	gpuErrchk(hipMemset(&(col_d->uncoloredNodes), false, sizeof(bool)));

	dim3 threads (THREADxBLOCK);
	dim3 blocks ((str->nodeSize + threads.x - 1) / threads.x, 1, 1 );

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord(start);

	while(col_h->uncoloredNodes){
		col_h->uncoloredNodes = false;
		col_h->numOfColors++;

		// Aggiorno coloring GPU
		gpuErrchk(hipMemcpy(&(col_d->uncoloredNodes), &(col_h->uncoloredNodes), sizeof(bool), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(&(col_d->numOfColors), &(col_h->numOfColors), sizeof(uint), hipMemcpyHostToDevice));

		colorer<<<blocks, threads>>>(col_d, str);

        //gpuErrchk(hipPeekAtLastError());
        //gpuErrchk(hipDeviceSynchronize());

		// Aggiorno uncoloredNodes lato CPU
		gpuErrchk(hipMemcpy(&(col_h->uncoloredNodes), &(col_d->uncoloredNodes), sizeof(bool), hipMemcpyDeviceToHost));
	}

    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tempo solo kernel %f ms\n", milliseconds);

	// Copio il risultato da CPU a GPU
	gpuErrchk(hipMemcpy(col_h->coloring, coloring_d, n * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&(col_h->numOfColors), &(col_d->numOfColors), sizeof(uint), hipMemcpyDeviceToHost));

    return col_h;
}